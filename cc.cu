#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <omp.h>
#include <vector>
#pragma GCC target("sse4,avx")
#pragma GCC optimize("unroll-loops")
#define getn S[df++]
char *S;
int *tempE,*rE,*c;

int df=0;
int *Li,*Ri,*dsu,*nex,*id,*linked_list,fill=0,n,m,*fin_table;
int db=-1, cpu_threads=4, gpu_threads=3;
int readInt()
{
	int m = 0;
	char g = getn;
	while (g<'0' || g>'9')
		g = getn;
	while (g <= '9'&& g >= '0')
	{
		m = (m << 1) + (m << 3) + g - 48;
		g = getn;
	}
	return m;
}
void countingSort(int *A) 
{ 
	int i, sum, maxi = max(n,m);
	memset(c, 0, (max(n,m))*sizeof(int)); 
	for (i = 0; i < m; i++) 
		c[A[i]]++;
	for (i = sum = 0; i < maxi; i++)
	{
		int t = c[i]; c[i] = sum; sum += t; 
	}
	for (i = 0; i < m; i++) 
		tempE[c[A[rE[i]]]++] = rE[i];
	for (i = 0; i < m; i++)
		rE[i] = tempE[i];
}
void sort_linear()
{
	rE=(int*)malloc(sizeof(int)*(max(m,n)));
	tempE=(int*)malloc(sizeof(int)*(max(m,n)));
	int z=n;
	if(m>z)
	z=m;
	c=(int*)malloc(sizeof(int)*(max(n,m)));
	for(int i=0;i<m;i++)
	rE[i]=i;
	countingSort(Ri);
	countingSort(Li);
	

	for(int i=0;i<m;i++)
	{
		c[i]=Li[rE[i]];
		tempE[i]=Ri[rE[i]];
	}
	for(int i=0;i<m;i++)
		Li[i]=c[i],Ri[i]=tempE[i];
}
int find_set(int L)
{
	if(dsu[L]==L)
		return L;
	 dsu[L]=find_set(dsu[L]);
	return dsu[L];
}
void unite(int p,int q)
{
	int a=find_set(p),b=find_set(q);
	if(a!=b)
		dsu[a]=b;
}
int is_connected(int a,int b)
{
	return find_set(a)==find_set(b);
}
int get_new()
{
	return fill++;
}
void set_val(int idx,int val)
{
	id[idx]=val;
}
void add(int node,int list_no)
{
	if(linked_list[list_no]==-1)
	{
		linked_list[list_no]=get_new();
		set_val(linked_list[list_no],node);
	}
	else
	{
		int N=get_new();
		nex[N]=linked_list[list_no];
		linked_list[list_no]=N;
		set_val(N,node);
	}
}
void read_input(char *file_name)
{
	df=0;
	FILE *f = fopen(file_name, "r");
	fseek(f, 0, SEEK_END);
	long fsize = ftell(f);
	fseek(f, 0, SEEK_SET); 
	S =(char*) malloc(fsize + 1);
	fread(S, fsize, 1, f);
	fclose(f);
	S[fsize] = 0;
	
}
 int find_set_serial(int L,int offset, int *dsu)
{
	int L0=L;
	while(dsu[L-offset]!=L)
	{
		L=dsu[L-offset];
	}
	while(dsu[L0-offset]!=L0)
	{
		int c=dsu[L0-offset];
		dsu[L0-offset]=L;
		L0=c;
	}
	return L0;
}
 void unite_serial(int p,int q,int offset, int *dsu)
{	

	int a=find_set_serial(p,offset,dsu),b=find_set_serial(q,offset,dsu);
	if(a!=b)
		dsu[a-offset]=b;

}
__device__ int find_set(int L,int offset, int *dsu)
{
	int L0=L;
	while(dsu[L-offset]!=L)
	{
		L=dsu[L-offset];
	}
	while(dsu[L0-offset]!=L0)
	{
		int c=dsu[L0-offset];
		dsu[L0-offset]=L;
		L0=c;
	}
	return L0;
}
 __device__ void unite(int p,int q,int offset, int *dsu)
{	

	int a=find_set(p,offset,dsu),b=find_set(q,offset,dsu);
	if(a!=b)
		dsu[a-offset]=b;

}
__global__ void connected_components( int *dsu, int *EL, int *ER,
		 int n, int m, int Li, int Ri) 
{
	 int ID = blockIdx.x;
	 int sz = gridDim.x;
	 int eli=((Ri-Li+1)/sz+1)*ID+Li;
	 if(eli>Ri)
		return;
	 int endi=eli+(Ri-Li+1)/sz;
	 if(endi>Ri)
		endi=Ri;
	int a=0,b=m-1;
	/*while(b-a>1)
	{
		int mid=(a+b)/2;
		if(EL[mid]<eli)
		{
			a=mid;
		}
		else
			b=mid;
	}
*/
	 for(int i=a;i<m;i++)
	{
		int li=EL[i],ri=ER[i];
		if(eli<=li && ri<=endi && li!=ri)
		{
			unite(li,ri,Li,dsu);
		}
	/*	if(li>endi)
			break;*/
	}
}
int *device_dsu,*dev_EL,*dev_ER;
void mem_init_cuda(int start_idx,int end_idx)
{
	hipMalloc((void **) &device_dsu, (end_idx-start_idx+1)*sizeof(int));
	hipMalloc((void **) &dev_EL, m*sizeof(int));
	hipMalloc((void **) &dev_ER, m*sizeof(int));
	
	hipMemcpy(device_dsu, dsu+start_idx, (end_idx-start_idx+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_EL, Li, m*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_ER, Ri, m*sizeof(int), hipMemcpyHostToDevice);
}
void mem_release_cuda(int start_idx,int end_idx)
{
	hipMemcpy(dsu+start_idx, device_dsu, (end_idx-start_idx+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(device_dsu);
	hipFree(dev_EL);
	hipFree(dev_ER);
}
void execute_on_gpu(int start_idx,int end_idx,int number_of_threads)
{
	connected_components<<<number_of_threads,1>>>(device_dsu,dev_EL,dev_ER,n,m,start_idx,end_idx);
	return ;
}
void execute_on_cpu(int start_idx,int end_idx,int number_of_threads)
{
int tid;
#pragma omp parallel for
 	for( tid=0;tid<number_of_threads;tid++)
	{
		int ID = tid;
		int sz = number_of_threads;
		int eli=((end_idx-start_idx+1)/sz+1)*ID+start_idx;
		 
		if(eli>end_idx)
			continue;
		int endi=eli+(end_idx-start_idx+1)/sz;
		
		if(endi>end_idx)
			endi=end_idx;
		int a=0,b=m-1;
		/* uncomment to turn on optimization
		while(b-a>1)
		{
			int mid=(a+b)/2;
			if(Li[mid]<eli)
			{
				a=mid;
			}
			else
				b=mid;
		}
		   */ 
		for(int i=a;i<m ;i++)
		{
			int li=Li[i],ri=Ri[i];
			if(eli<=li && ri<=endi && li!=ri)
			{
				unite_serial(li,ri,start_idx,dsu);
			}
		}
	}
}
void finish(int l,int z1,int z2);
int device_info();
void make_table(int,int,int);
int main(int argc,char** argv)
{
	/*
	argv[1]=gpu_threads
	argv[2]=cpu_threads
	argv[3]= input file 
	argv[4]= output file
	argv[5]= logfile
	input file format
	A single line of input first integer 'n'
	second integer 'm' then m pairs of integers denoting edges
	output : Connected components in each line
	*/
	gpu_threads=4096;
	cpu_threads=omp_get_max_threads();
		if(argc>1)	
	gpu_threads=atoi(argv[1]);
	if(argc>2)
		cpu_threads=atoi(argv[2]);
	char file[] = "input.txt";
	char file2[] = "Pdsuout.txt";
	char file3[] = "Pdsulog.txt";
	if (argc>3)
		strcpy(file,argv[3]);
	if(argc>4)
		strcpy(file2,argv[4]);
	if(argc>5)
		strcpy(file3, argv[5]);
	clock_t Input_time, Comp_time, Tot_exec_time, Output_time;
	Input_time = clock();
	read_input(file);
	Tot_exec_time = Input_time;
	Comp_time=clock();
	Input_time=Comp_time - Input_time;
	 n=readInt(), m=readInt();
	dsu=(int*)malloc(sizeof(int)*n);
	nex=(int*)malloc(sizeof(int)*n);
	id=(int*)malloc(sizeof(int)*n);
	fin_table=(int*)malloc(sizeof(int)*n);
	Li=(int*)malloc(sizeof(int)*m);
	Ri=(int*)malloc(sizeof(int)*m);
	linked_list=(int*)malloc(sizeof(int)*n);
	memset((void*)linked_list,255,n*sizeof(int));
	memset((void*)nex,255,n*sizeof(int));
	for(int i=0;i<n;i++)
		dsu[i]=i;
	for(int i=0;i<m;i++)
	{
		Li[i]=readInt()-1;
		Ri[i]=readInt()-1;
		if(Li[i]>Ri[i])
		{
		// swapping values
			Li[i]^=Ri[i];
			Ri[i]^=Li[i];
			Li[i]^=Ri[i];
		}
	}
		
//	sort_linear();
fprintf(stderr,"ok %d %d",n,m);
	db=n/3;
	mem_init_cuda(db+1,n-1);
	make_table(db,cpu_threads,gpu_threads);
	Comp_time=clock();
	execute_on_gpu(db+1, n - 1, gpu_threads);
	execute_on_cpu(0,db, cpu_threads);
	hipDeviceSynchronize();
	mem_release_cuda(db+1,n-1);
	finish(db,cpu_threads,gpu_threads);
	Output_time=clock();
	mem_release_cuda(db+1,n-1);
	for(int i=0;i<n;i++)
	{
		add(i,find_set(i));
	}
	FILE* fout=fopen(file2,"w");
	Comp_time=Output_time-Comp_time;
	for(int i=0;i<n;i++)
	{
		int f=linked_list[i];
		if(f!=-1)
		{
			while(f!=-1)
			{
				fprintf(fout,"%d ",id[f]+1);
				f=nex[f];
			}
			fprintf(fout,"\n");
		}
	}
	fclose(fout);
	clock_t temp=clock();
	Output_time=temp-Output_time;
	Tot_exec_time=temp-Tot_exec_time;
	FILE* fst=fopen(file3,"a");
	fprintf(fst,"-------------------------------------------------------------\n");
	time_t end_time;
	time(&end_time);
	fprintf(fst, "Finished computation at : %s", ctime(&end_time));
	fprintf(fst, "Nodes : %d, Edges : %d\n",n,m);
	fprintf(fst,"Input time = %.3fms\n",(Input_time*1.0/CLOCKS_PER_SEC)*1000.0);
	fprintf(fst,"Computation time = %.3fms\n",(Comp_time*1.0/CLOCKS_PER_SEC)*1000.0);
	fprintf(fst,"Output time = %.3fms\n",(Output_time*1.0/CLOCKS_PER_SEC)*1000.0);
	fprintf(fst,"Total execution time = %.3fms\n",(Tot_exec_time*1.0/CLOCKS_PER_SEC)*1000.0);
	fclose(fst);	
	return 0;
}
void make_table(int l,int z1,int z2)
{
	int ft = (l + 1) / z1 + 1, ft2 = (n - 1 - l) / z2 + 1;
	int l0=0,j=0,l1=0;
	while(j<=l)
	{
		fin_table[j]=l1;
		++l0;
		if(l0>=ft)
		{
			l0=0;
			l1++;
		}
		++j;
	}
	j=l+1;
	l1=0;
	l0=0;
	while(j<n)
	{
		fin_table[j]=l1;
		++l0;
		if(l0>=ft2)
		{
			l0=0;
			l1++;
		}
		++j;
	}
}
void finish(int l, int z1, int z2)
{
	// first cpu then gpu nodes
	// z1,z2 cpu and gpu threads
	
	for (int i = 0; i < m; i++)
	{
		if (Ri[i] <= l)
		{
			if (fin_table[Li[i]] != fin_table[Ri[i]])
				unite(Li[i], Ri[i]);
		}
		else if(Li[i]>l)
		{
			if (fin_table[Li[i]]!= fin_table[Ri[i]])
				unite(Li[i], Ri[i]);
		}
	//	else
			unite(Li[i], Ri[i]);
	}
}
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %u\n",  devProp.totalConstMem);
    printf("Texture alignment:             %u\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}
 
int device_info()
{
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);
 
    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    } 
    return 0;
}
